
#include <hip/hip_runtime.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <sstream>
#include <stdlib.h>
#include <string>

using namespace std;

#define Sq(x) ((x) * (x))

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__device__ double delt_d;
__device__ double freeze_temp_d;

__device__ double briq_temp_d;
__device__ double goo_temp_d;
__device__ double bound_temp_d;

__device__ int sphere_count_x_d;
__device__ int sphere_count_y_d;
__device__ int sphere_count_z_d;

__device__ int problem_size_x_d;
__device__ int problem_size_y_d;
__device__ int problem_size_z_d;

__device__ double dx_d;
__device__ double dt_d;

__device__ int get_pos(int i, int j, int k)
{
    return i + j * problem_size_x_d + k * problem_size_x_d * problem_size_y_d;
}

__device__ double c_ro(double t)
{
    double c_frosen = 1600;
    double ro_frosen = 1200;

    double c_melt = 1900;
    double ro_melt = 1200;

    double L = 330000 * 0.1;

    if (t < freeze_temp_d - delt_d)
        return c_frosen * ro_frosen;

    if (t >= freeze_temp_d - delt_d && t < freeze_temp_d)
        return (c_frosen + L / delt_d / 2) * ro_frosen;

    if (t >= freeze_temp_d && t < freeze_temp_d + delt_d)
        return (c_melt + L / delt_d / 2) * ro_melt;

    if (t >= freeze_temp_d + delt_d)
        return c_melt * ro_melt;

    return 0;
}

__device__ double k(double t)
{
    double k_frosen = 0.92;

    double k_melt = 0.72;

    double scale = 3600;

    if (t < freeze_temp_d)
        return k_frosen * scale;

    if (t >= freeze_temp_d)
        return k_melt * scale;

    return 0;
}

__global__ void init(double *in, double *out)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int idz = threadIdx.z + blockDim.z * blockIdx.z;

    double R = problem_size_x_d / sphere_count_x_d / 2.0;

    if (idx < problem_size_x_d && idy < problem_size_y_d && idz < problem_size_z_d) {
        in[get_pos(idx, idy, idz)] = briq_temp_d;
        out[get_pos(idx, idy, idz)] = briq_temp_d;
    }

    if (idx > 0 && idx < problem_size_x_d - 1 && idy > 0 && idy < problem_size_y_d - 1 && idz > 0 &&
        idz < problem_size_z_d - 1) {
        in[get_pos(idx, idy, idz)] = goo_temp_d;
        out[get_pos(idx, idy, idz)] = goo_temp_d;
        for (int i = 0; i < sphere_count_x_d; i++)
            for (int j = 0; j < sphere_count_y_d; j++)
                for (int k = 0; k < sphere_count_z_d; k++)
                    if (Sq(idx - R * (1 + 2 * i)) + Sq(idy - R * (1 + 2 * j)) +
                            Sq(idz - R * (1 + 2 * k)) <=
                        Sq(R + 1)) {
                        in[get_pos(idx, idy, idz)] = briq_temp_d;
                        out[get_pos(idx, idy, idz)] = briq_temp_d;
                    }
    }
}

__global__ void solve(double *out, double *in)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int idz = threadIdx.z + blockDim.z * blockIdx.z;

    if (idx > 0 && idx < problem_size_x_d - 1 && idy > 0 && idy < problem_size_y_d - 1 && idz > 0 &&
        idz < problem_size_z_d - 1) {
        if (idx == 1) {
            in[get_pos(idx - 1, idy, idz)] = bound_temp_d; //in[get_pos(idx, idy, idz)];
        } else if (idx == problem_size_x_d - 2) {
            in[get_pos(idx + 1, idy, idz)] = bound_temp_d; //in[get_pos(idx, idy, idz)];
        } else if (idy == 1) {
            in[get_pos(idx, idy - 1, idz)] = in[get_pos(idx, idy, idz)];
        } else if (idy == problem_size_y_d - 2) {
            in[get_pos(idx, idy + 1, idz)] = in[get_pos(idx, idy, idz)];
        } else if (idz == 1) {
            in[get_pos(idx, idy, idz - 1)] = in[get_pos(idx, idy, idz)];
        } else if (idz == problem_size_z_d - 2) {
            in[get_pos(idx, idy, idz + 1)] = in[get_pos(idx, idy, idz)];
        }
        out[get_pos(idx, idy, idz)] =
            in[get_pos(idx, idy, idz)] +
            dt_d / ((dx_d * dx_d) * c_ro(in[get_pos(idx, idy, idz)])) *
                ((((k(in[get_pos(idx + 1, idy, idz)]) + k(in[get_pos(idx, idy, idz)])) / 2.0) *
                      (in[get_pos(idx + 1, idy, idz)] - in[get_pos(idx, idy, idz)]) -
                  ((k(in[get_pos(idx, idy, idz)]) + k(in[get_pos(idx - 1, idy, idz)])) / 2.0) *
                      (in[get_pos(idx, idy, idz)] - in[get_pos(idx - 1, idy, idz)])) +
                 (((k(in[get_pos(idx, idy + 1, idz)]) + k(in[get_pos(idx, idy, idz)])) / 2.0) *
                      (in[get_pos(idx, idy + 1, idz)] - in[get_pos(idx, idy, idz)]) -
                  ((k(in[get_pos(idx, idy, idz)]) + k(in[get_pos(idx, idy - 1, idz)])) / 2.0) *
                      (in[get_pos(idx, idy, idz)] - in[get_pos(idx, idy - 1, idz)])) +
                 (((k(in[get_pos(idx, idy, idz + 1)]) + k(in[get_pos(idx, idy, idz)])) / 2.0) *
                      (in[get_pos(idx, idy, idz + 1)] - in[get_pos(idx, idy, idz)]) -
                  ((k(in[get_pos(idx, idy, idz)]) + k(in[get_pos(idx, idy, idz - 1)])) / 2.0) *
                      (in[get_pos(idx, idy, idz)] - in[get_pos(idx, idy, idz - 1)])));
    }
}

static void swap4(float *v)
{
    char in[4], out[4];
    memcpy(in, v, 4);
    out[0] = in[3];
    out[1] = in[2];
    out[2] = in[1];
    out[3] = in[0];
    memcpy(v, out, 4);
}

int main()
{
    int max_size_x = 500;
    int max_size_y = 500;
    int max_size_z = 500;

    double dx = 0.01;
    double dt = 0.001;
    double delt = 0.1;

    double freeze_temp = 0;

    int sphere_count_x = 8;
    int sphere_count_y = 8;
    int sphere_count_z = 8;

    double centre_line[500];

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(sphere_count_x_d), &sphere_count_x, sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(sphere_count_y_d), &sphere_count_y, sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(sphere_count_z_d), &sphere_count_z, sizeof(int)));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dt_d), &dt, sizeof(double)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dx_d), &dx, sizeof(double)));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(freeze_temp_d), &freeze_temp, sizeof(double)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(delt_d), &delt, sizeof(double)));

    double *heat_array_old;
    gpuErrchk(hipMalloc((void **)&heat_array_old,
                         max_size_x * max_size_y * max_size_z * sizeof(double)));

    double *heat_array_now;
    gpuErrchk(hipMalloc((void **)&heat_array_now,
                         max_size_x * max_size_y * max_size_z * sizeof(double)));

    double *heat_array_cpu =
        (double *)malloc(max_size_x * max_size_y * max_size_z * sizeof(double));

    ofstream file_out("out.txt");

    for (int i = -30; i <= -30; i += 5) { //Temperature degrees
        for (int j = 30; j <= 30; j += 5) { //Size cm
            for (int k = 10; k <= 10; k += 5) { //Pulp temperature
                for (int d = -6; d <= -6; d += 3) { //Wall temperature
                    sphere_count_x = ceil(200.0 / j);
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(sphere_count_x_d), &sphere_count_x, sizeof(int)));

                    int problem_size_x = sphere_count_x * j * 0.01 / dx + 1;
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(problem_size_x_d), &problem_size_x, sizeof(int)));

                    int problem_size_y = sphere_count_y * j * 0.01 / dx + 1;
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(problem_size_y_d), &problem_size_y, sizeof(int)));

                    int problem_size_z = sphere_count_z * j * 0.01 / dx + 1;
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(problem_size_z_d), &problem_size_z, sizeof(int)));

                    double briq_temp = i;
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(briq_temp_d), &briq_temp, sizeof(double)));

                    double goo_temp = k;
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(goo_temp_d), &goo_temp, sizeof(double)));

                    double bound_temp = d;
                    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bound_temp_d), &bound_temp, sizeof(double)));

                    dim3 threadsPerBlock(8, 8, 8);
                    dim3 numBlocks(problem_size_x / threadsPerBlock.x + 1,
                                   problem_size_y / threadsPerBlock.y + 1,
                                   problem_size_z / threadsPerBlock.z + 1);

                    file_out << "Briquette temperature:" << briq_temp << "C "
                             << "Briquette side size:" << j * 0.01 << "m "
                             << "Pulp temperature:" << goo_temp << "C "
                             << "Wall temperature:" << bound_temp << "C" << endl
                             << "Time(clock)    Maximum pulp temperature(C)" << endl;

                    init<<<numBlocks, threadsPerBlock>>>(heat_array_old, heat_array_now);

                    for (int time = 0; time <= 168 / dt; time++) {
                        solve<<<numBlocks, threadsPerBlock>>>(heat_array_now, heat_array_old);

                        double *tmp = heat_array_now;
                        heat_array_now = heat_array_old;
                        heat_array_old = tmp;

                        if (time % ((int)(0.1 / dt)) == 0) {
                            gpuErrchk(hipMemcpy(
                                &centre_line,
                                &heat_array_old[0 + problem_size_y / 2 * problem_size_x +
                                                problem_size_z / 2 * problem_size_x *
                                                    problem_size_y],
                                problem_size_x * sizeof(double), hipMemcpyDeviceToHost));
                            double max_value = centre_line[0];
                            for (int i = 0; i < problem_size_x; i++) {
                                if (max_value < centre_line[i])
                                    max_value = centre_line[i];
                            }
                            file_out << time * dt << "    " << max_value << endl;

                            gpuErrchk(hipMemcpy(heat_array_cpu, heat_array_old,
                                                 problem_size_x * problem_size_y * problem_size_z *
                                                     sizeof(double),
                                                 hipMemcpyDeviceToHost));
                            char out_string[100];
                            sprintf(out_string, "plot/result_%d.vtk", time);
                            ofstream out(out_string, ios::out | ios::binary);
                            out << "# vtk DataFile Version 2.0" << endl;
                            out << "Heat" << endl;
                            out << "BINARY" << endl;
                            out << "DATASET STRUCTURED_POINTS" << endl;
                            out << "DIMENSIONS " << problem_size_x << " " << problem_size_y << " "
                                << problem_size_z << endl;
                            out << "ASPECT_RATIO 1 1 1" << endl;
                            out << "ORIGIN 0 0 0" << endl;
                            out << "POINT_DATA " << problem_size_x * problem_size_y * problem_size_z
                                << endl;
                            out << "SCALARS heat float 1" << endl;
                            out << "LOOKUP_TABLE default" << endl;
                            for (int k = 0; k < problem_size_z; k++) {
                                for (int j = 0; j < problem_size_y; j++) {
                                    for (int i = 0; i < problem_size_x; i++) {
                                        float tmp =
                                            heat_array_cpu[i + j * problem_size_x +
                                                           k * problem_size_x * problem_size_y];
                                        swap4(&tmp);
                                        out.write((char *)(&tmp), sizeof(float));
                                    }
                                }
                            }
                            out << endl;
                            out.close();

                            if (max_value < -0.1)
                                break;
                        }
                    }

                    file_out << endl;
                }
            }
        }
    }

    return 0;
}
